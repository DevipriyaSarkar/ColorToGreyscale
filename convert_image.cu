#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int tx = threadIdx.x + blockIdx.x * blockDim.x;
  int ty = threadIdx.y + blockIdx.y * blockDim.y;
  uchar4 image = rgbaImage[tx + ty * numCols];
  float R = image.x;
  float G = image.y;
  float B = image.z;
  float A = image.w;
  greyImage[tx + ty * numCols] = 0.299f * R + 0.587f * G + 0.114f * B;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int block_width = 16;
  int block_height = 16;
  int grid_width = numCols / block_width;
  if (grid_width * block_width < numCols)
      grid_width++;
  int grid_height = numRows / block_height;
  if (grid_height * block_height < numRows)
      grid_height++;
  
  const dim3 blockSize(block_width, block_height );  //TODO
  const dim3 gridSize( grid_width, grid_height );  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
